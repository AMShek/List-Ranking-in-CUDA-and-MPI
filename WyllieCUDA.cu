#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>

int * gen_linked_list_1(int N);
int* gen_linked_list_2(int N);

__global__ void ListRank
(int* List, int size)
{
	int block = (blockIdx.y*gridDim.x) + blockIdx.x;
	int index = block * blockDim.x + threadIdx.x;

	if (index<size) {
		while (1) {
			int node = LIST[index];
			if (node >> 32 == -1) return;
			__syncthreads();

			int mask = 0xFFFFFFFF;
			ing temp = 0;
			int next = LIST[node >> 32];

			if (node >> 32 == -1) return;

			temp = node & mask;
			temp += next & mask;
			temp += (next >> 32) << 32;

			__syncthreads();
			LIST[index] = temp;
		}
	}
}


int main() {

	int N = 10;
	mem_size_list = N * sizeof(int);

	//get the array of values in list
	int* listptr = NULL;
	listptr = gen_linked_list_1(N);

	//array to store rank
	int* rank = NULL;
	rank = (int*)malloc(N * sizeof(int));

	hipMalloc(&d_list, mem_size_list);
	hipMemcpy(d_list, listptr, mem_size_list, hipMemcpyHostToDevice);

	//setup the execution configuration
	int dimGrid = 10;
	int dimBlock = 1000;

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	hipEventCreate(&start);

	hipEvent_t stop;
	hipEventCreate(&stop);

	// Record the start event
	hipEventRecord(start, NULL);

	ListRank << <dimGrid, dimBlock >> > (d_list, N);

	// Record the stop event
	hipEventRecord(stop, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);

	// Compute and print the performance
	printf("Time= %.3f msec\n", msecTotal);

	//Read C from device
	hipMemcpy(rank, d_list, mem_size_list, hipMemcpyDeviceToHost);

	hipFree(d_list);
	return 0;
}

int * gen_linked_list_1(int N)
{

	int * list = NULL;
	if (NULL != list)
	{
		free(list);
		list = NULL;
	}

	if (0 == N)
	{
		printf("N is 0, exit\n");
		exit(-1);
	}

	list = (int*)malloc(N * sizeof(int));
	if (NULL == list)
	{
		printf("Can not allocate memory for output array\n");
		exit(-1);
	}

	int i;
	for (i = 0; i<N; i++)
		list[i] = i - 1;

	return list;
}

int* gen_linked_list_2(int N)
{
	int * list;

	list = gen_linked_list_1(N);

	int p = N / 5;

	int i, temp;

	for (i = 0; i<N; i += 2)
	{
		temp = list[i];
		list[i] = list[(i + (i + p)) % N];
		list[(i + (i + p)) % N] = temp;
	}

	return list;
}